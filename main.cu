#include <iostream>
#include <SDL2/SDL.h>
#include <SDL2/SDL_image.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

namespace asteroids{

using vec2f = std::vector<float, float>;

SDL_Texture *m_texture;

void render(){

}

int main(int argc, char** argv)
{
	SDL_Event e;
	const Uint8* currentKeyStates;
	bool quit = false;

	SDL_Init(SDL_INIT_VIDEO);
	SDL_Window *window;
	SDL_Renderer *renderer;
	window = SDL_CreateWindow("Asteroids", 0, 0, 800, 600, SDL_WINDOW_SHOWN);
	renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
	
	SDL_Surface *surface = IMG_Load(argv[1]);
        m_texture = SDL_CreateTextureFromSurface(renderer, surface);


	while(!quit){
		while(SDL_PollEvent(&e)){
			if(e.type == SDL_QUIT){
				quit = true;
			}
		}
		SDL_RenderClear(renderer);
		render();
	}
	std::cout << "Hello, World!" << std::endl;
	return 0;
}


struct asteroid{
	std::vector<float> pos;
	float mass;

};


}
