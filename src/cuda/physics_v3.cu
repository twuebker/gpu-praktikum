#include "hip/hip_runtime.h"
#define EPS (10)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>
#include <thrust/device_vector.h>
#include <stdio.h>

#define blockSize  16
#define blockFloat 16.0

void __global__ calculate_forces_v3(Asteroid* d_a, float dt, int size, ForceField* d_f,
        int sizeFF, float2* accs)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    Asteroid* asteroidI;
    Asteroid* asteroidJ;
    if(i < size && j < size) {
	   
    	asteroidI = &d_a[i];
    	asteroidJ = &d_a[j];

    float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
    float2 acc = {0, 0}; //Acceleration in x and y direction

    if (i!=j)
    {

        float2 distance_dir = {asteroidI->pos.first-asteroidJ->pos.first, asteroidI->pos.second-asteroidJ->pos.second};
        float distance = sqrt(
                distance_dir.x*distance_dir.x+distance_dir.y*distance_dir.y); //Pythagoras r = sqrt(x² + y²)

        // Bei Distanz gegen 0 geht Kraft gegen unendlich
        // -> Kraft begrenzen durch Softening Faktor EPS
        float soften_dist = sqrt(distance*distance+EPS*EPS);
        float acceleration = -1.0*BIG_G*asteroidJ->mass/(soften_dist*soften_dist);
        float2 distance_dir_unit = {distance_dir.x/soften_dist,
                                    distance_dir.y/soften_dist}; //Distance Vector normalisiert (durch Länge geteilt)

        acc.x += acceleration*distance_dir_unit.x;
        acc.y += acceleration*distance_dir_unit.y;

    }
    else
    {

        for (int k = 0; k<sizeFF; k++)
        {
            ForceField force = d_f[k];
            if (!(asteroidI->pos.first>=force.leftCorner.first && asteroidI->pos.first<=force.rightCorner.first
                    && asteroidI->pos.second>=force.leftCorner.second
                    && asteroidI->pos.second<=force.rightCorner.second))
            {
                continue;
            }
            switch (force.dir)
            {
            case Direction::UP: acc.y -= force.force/asteroidI->mass;
                break;
            case Direction::DOWN: acc.y += force.force/asteroidI->mass;
                break;
            case Direction::LEFT: acc.x -= force.force/asteroidI->mass;
                break;
            case Direction::RIGHT: acc.x += force.force/asteroidI->mass;
                break;
            default: break;
            }
        }
    }
    accs[threadIdx.y * blockDim.x + threadIdx.x].x = acc.x;
    accs[threadIdx.y * blockDim.x + threadIdx.x].y = acc.y;
}
    __syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
    //die späteren Blocke mit veränderten Daten arbeiten
    // Size 5: Threads 0 - 4, shared arr size 25, i,j zwischen 0 - 4

    for (unsigned int s = 1; s<blockDim.x; s *= 2)
    {
        if (threadIdx.y%(2*s)==0 && i < size && j < size && blockIdx.y * blockDim.y + (threadIdx.y + s) < size)
        {
            accs[threadIdx.y* blockDim.x + threadIdx.x].x += accs[(threadIdx.y+s)*blockDim.x + threadIdx.x].x;
            accs[threadIdx.y* blockDim.x + threadIdx.x].y += accs[(threadIdx.y+s)*blockDim.x + threadIdx.x].y;
        }
        __syncthreads();
    }

    if (threadIdx.x == threadIdx.y && i < size && j < size)
    {
        Asteroid* ast = &d_a[i];
        ast->velocity.first += accs[threadIdx.x].x*dt;
        ast->velocity.second += accs[threadIdx.x].y*dt;
    }


}

void __global__ updatePositions(Asteroid* d_asteroid, int size, float dt) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size) {
		Asteroid* ast = &d_asteroid[i];
		ast->pos.first += ast->velocity.first * dt;
		ast->pos.second += ast->velocity.second * dt;
	}
}


void freeDeviceMemory_v3(Asteroid* d_asteroid, ForceField* d_forceField, float2* d_accs)
{
    hipFree(d_asteroid);
    hipFree(d_forceField);
    hipFree(d_accs);
}

std::tuple<Asteroid*, ForceField*, float2*> updateMemory_v3(std::vector<Asteroid>& asteroids, std::vector<ForceField>& forceFields)
{
    Asteroid* d_asteroid;
    ForceField* d_forceField;
    float2* d_accs;
    int size = sizeof(Asteroid)*asteroids.size();
    int sizeFF = sizeof(ForceField)*forceFields.size();
    int sizeAccs = sizeof(float2)* blockSize * blockSize;
    auto start = std::chrono::steady_clock::now();
    hipMalloc(&d_asteroid, size);
    hipMalloc(&d_forceField, sizeFF);
    hipMalloc(&d_accs, sizeAccs);
    hipMemcpy(d_asteroid, asteroids.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_forceField, forceFields.data(), sizeFF, hipMemcpyHostToDevice);
    auto end = std::chrono::steady_clock::now();
    float dif = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    MessSaver::add("GPUV3 Hinkopieren", dif);
    return std::make_tuple(d_asteroid, d_forceField, d_accs);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code!=hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void call_kernel_v3(Asteroid* a, Asteroid* d_asteroid, ForceField* d_forceField, int sizeAsteroids, int sizeForceFields, float2* d_accs)
{
    if (sizeAsteroids==0)
    {
        return;
    }
	dim3 dim(std::ceil(sizeAsteroids / blockFloat), std::ceil(sizeAsteroids / blockFloat));
	auto start = std::chrono::steady_clock::now();

    calculate_forces_v3<<<dim, dim3(blockSize, blockSize)>>>(d_asteroid, 0.1, sizeAsteroids,
            d_forceField, sizeForceFields, d_accs);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    updatePositions<<<std::ceil(sizeAsteroids / blockFloat * blockFloat), blockSize * blockSize>>>(d_asteroid, sizeAsteroids, 0.1);	
	auto end = std::chrono::steady_clock::now();
    float dif = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	 MessSaver::add("GPUV3 Berechnen", dif);
     start = std::chrono::steady_clock::now();
    hipMemcpy(a, d_asteroid, sizeof(Asteroid)*sizeAsteroids, hipMemcpyDeviceToHost);
    end = std::chrono::steady_clock::now();
    dif = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
         MessSaver::add("GPUV3 Zurückkopieren", dif);
}

