#include "hip/hip_runtime.h"
#define EPS (10)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>
#include <thrust/device_vector.h>
#include <stdio.h>

void __global__ calculate_forces_v3(Asteroid* d_a, float dt, int size, ForceField* d_f,
        int sizeFF)    //DelteTime ist die Zeit die zwischen zwei Berechnungen vergeht. Mussen irgendwie
{
    extern __shared__ float2 accs[];
    printf("IM KERNEL\n");
    int i = threadIdx.x;
    int j = threadIdx.y;
    Asteroid* asteroidI = &d_a[i];
    Asteroid* asteroidJ = &d_a[j];

    float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
    float2 acc = {0, 0}; //Acceleration in x and y direction

    if (i!=j)
    {

        float2 distance_dir = {asteroidI->pos.first-asteroidJ->pos.first, asteroidI->pos.second-asteroidJ->pos.second};
        float distance = sqrt(
                distance_dir.x*distance_dir.x+distance_dir.y*distance_dir.y); //Pythagoras r = sqrt(x² + y²)

        // Bei Distanz gegen 0 geht Kraft gegen unendlich
        // -> Kraft begrenzen durch Softening Faktor EPS
        float soften_dist = sqrt(distance*distance+EPS*EPS);
        float acceleration = -1.0*BIG_G*asteroidJ->mass/(soften_dist*soften_dist);
        float2 distance_dir_unit = {distance_dir.x/soften_dist,
                                    distance_dir.y/soften_dist}; //Distance Vector normalisiert (durch Länge geteilt)

        acc.x += acceleration*distance_dir_unit.x;
        acc.y += acceleration*distance_dir_unit.y;

    }
    else
    {

        for (int k = 0; k<sizeFF; k++)
        {
            ForceField force = d_f[k];
            if (!(asteroidI->pos.first>=force.leftCorner.first && asteroidI->pos.first<=force.rightCorner.first
                    && asteroidI->pos.second>=force.leftCorner.second
                    && asteroidI->pos.second<=force.rightCorner.second))
            {
                continue;
            }
            switch (force.dir)
            {
            case Direction::UP: acc.y -= force.force/asteroidI->mass;
                break;
            case Direction::DOWN: acc.y += force.force/asteroidI->mass;
                break;
            case Direction::LEFT: acc.x -= force.force/asteroidI->mass;
                break;
            case Direction::RIGHT: acc.x += force.force/asteroidI->mass;
                break;
            default: break;
            }
        }
    }
    accs[j*size+i].x = acc.x;
    accs[j*size+i].y = acc.y;

    __syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
    //die späteren Blocke mit veränderten Daten arbeiten
    for (unsigned int s = 1; s<size; s *= 2)
    {
        if (j%(2*s)==0)
        {
            accs[j*size+i].x += accs[(j+s)*size+i].x;
            accs[j*size+i].y += accs[(j+s)*size+i].y;
        }
        __syncthreads();
    }

    if (i==j)
    {
        Asteroid* ast = &d_a[i];
        ast->velocity.first += accs[i].x*dt;
        ast->velocity.second += accs[i].y*dt;

        ast->pos.first += ast->velocity.first*dt;
        ast->pos.second += ast->velocity.second*dt;
        if (i==1)
        {
            printf("adding %f, %f to ast 1\n", ast->velocity.first, ast->velocity.second);
        }

    }

}

void freeDeviceMemory(Asteroid* d_asteroid, ForceField* d_forceField)
{
    hipFree(d_asteroid);
    hipFree(d_forceField);
}

std::pair<Asteroid*, ForceField*> updateMemory(std::vector<Asteroid>& asteroids, std::vector<ForceField>& forceFields)
{
    Asteroid* d_asteroid;
    ForceField* d_forceField;
    int size = sizeof(Asteroid)*asteroids.size();
    int sizeFF = sizeof(ForceField)*forceFields.size();
    hipMalloc(&d_asteroid, size);
    hipMalloc(&d_forceField, sizeFF);
    hipMemcpy(d_asteroid, asteroids.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_forceField, forceFields.data(), sizeFF, hipMemcpyHostToDevice);
    return std::make_pair(d_asteroid, d_forceField);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code!=hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void call_kernel_v3(Asteroid* a, Asteroid* d_asteroid, ForceField* d_forceField, int sizeAsteroids, int sizeForceFields)
{
    if (sizeAsteroids==0)
    {
        return;
    }
    std::cout << "Kernel invoked" << std::endl;
    int sharedArraySize = sizeof(float2)*sizeAsteroids*sizeAsteroids;
    calculate_forces_v3<<<1, dim3(sizeAsteroids, sizeAsteroids), sharedArraySize>>>(d_asteroid, 0.1, sizeAsteroids,
            d_forceField, sizeForceFields);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipMemcpy(a, d_asteroid, sizeof(Asteroid)*sizeAsteroids, hipMemcpyDeviceToHost);
    if (sizeAsteroids>=2)
    {
        std::cout << "Pos: " << a[1].pos.first << "," << a[1].pos.second << std::endl;
    }
}
