#include "hip/hip_runtime.h"
#define EPS (10)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>
#include <thrust/device_vector.h>
#include <stdio.h>

void __global__ calculate_forces_v2(Asteroid* d_a, float dt, int size, ForceField* d_f, int sizeFF) 	//DelteTime ist die Zeit die zwischen zwei Berechnungen vergeht. Mussen irgendwie
{	
	//in der MainLoop dafür die Zeit stoppen und hier übergeben
	float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
	int astId = blockIdx.x * blockDim.x + threadIdx.x;
	Asteroid* ast = &d_a[astId];
	float2 acc = {0,0}; //Acceleration in x and y direction

	for(int i = 0; i < size; i++){
		if(i == astId) continue; //Keine Gravity zu sich selber

		Asteroid other = d_a[i];
		float2 distance_dir = {ast->pos.first - other.pos.first, ast->pos.second - other.pos.second};
		float distance = sqrt(distance_dir.x * distance_dir.x + distance_dir.y * distance_dir.y); //Pythagoras r = sqrt(x² + y²)
		
		// Bei Distanz gegen 0 geht Kraft gegen unendlich
		// -> Kraft begrenzen durch Softening Faktor EPS
		float soften_dist = sqrt(distance * distance + EPS * EPS);
		float acceleration = -1.0 * BIG_G * other.mass / (soften_dist * soften_dist);
	       	
		float2 distance_dir_unit = {distance_dir.x / soften_dist, distance_dir.y / soften_dist}; //Distance Vector normalisiert (durch Länge geteilt)

		acc.x += acceleration * distance_dir_unit.x;
		acc.y += acceleration * distance_dir_unit.y;	
	}
	
	for(int i = 0; i < sizeFF; i++) {
		ForceField force = d_f[i];
		if(!(ast->pos.first >= force.leftCorner.first && ast->pos.first <= force.rightCorner.first && ast->pos.second >= force.leftCorner.second && ast->pos.second <= force.rightCorner.second)) {
			continue;
		}
		switch(force.dir) {
				case Direction::UP:
						acc.y -= force.force /  ast->mass;
						break;
				case Direction::DOWN:
						acc.y += force.force / ast->mass;
						break;
				case Direction::LEFT:
						acc.x -= force.force / ast->mass;
						break;
				case Direction::RIGHT:
						acc.x += force.force / ast-> mass;		
						break;
				default:
					break;		
		}
	}
	__syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
                        //die späteren Blocke mit veränderten Daten arbeiten


	ast->velocity.first += acc.x * dt;
	ast->velocity.second += acc.y * dt;
	
	printf("pos: %f, %f\n", ast->pos.first, ast->pos.second);
	printf("wert der draufkommt: %f, %f\n", ast->velocity.first * dt, ast->velocity.second * dt);
	ast->pos.first += ast->velocity.first * dt;
	ast->pos.second += ast->velocity.second * dt;

}	

void freeDeviceMemory(Asteroid* d_asteroid, ForceField* d_forceField) {
	hipFree(d_asteroid);
	hipFree(d_forceField);
}

std::pair<Asteroid*, ForceField*> updateMemory(std::vector<Asteroid>& asteroids, std::vector<ForceField>& forceFields) {
	Asteroid* d_asteroid;
	ForceField* d_forceField;
	int size = sizeof(Asteroid) * asteroids.size();
	int sizeFF = sizeof(ForceField) * forceFields.size();
	hipMalloc(&d_asteroid, size);
	hipMalloc(&d_forceField, sizeFF);
	hipMemcpy(d_asteroid, asteroids.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_forceField, forceFields.data(), sizeFF, hipMemcpyHostToDevice);
	return std::make_pair(d_asteroid, d_forceField);
}

void call_kernel_v2(Asteroid* a, Asteroid* d_asteroid, ForceField* d_forceField, int sizeAsteroids, int sizeForceFields)
{
	if(sizeAsteroids == 0) {
		return;
	}
	calculate_forces_v2<<<1, sizeAsteroids>>>(d_asteroid, 0.1, sizeAsteroids, d_forceField, sizeForceFields);
	hipMemcpy(a, d_asteroid, sizeof(Asteroid) * sizeAsteroids, hipMemcpyDeviceToHost);
	std::cout << "Pos nach dem kernel: " << a[0].pos.first << "," << a[0].pos.second << std::endl;
}
