#include "hip/hip_runtime.h"
#define EPS (10)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"
#include <thrust/device_vector.h>
#include <stdio.h>

void __global__ calculate_forces(Asteroid* d_a, float dt, int size) 	//DelteTime ist die Zeit die zwischen zwei Berechnungen vergeht. Mussen irgendwie
{	
	//in der MainLoop dafür die Zeit stoppen und hier übergeben
	float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
	int astId = blockIdx.x * blockDim.x + threadIdx.x;
	Asteroid* ast = &d_a[astId];
	float2 acc = {0,0}; //Acceleration in x and y direction

	for(int i = 0; i < size; i++){
		if(i == astId) continue; //Keine Gravity zu sich selber

		Asteroid other = d_a[i];
		float2 distance_dir = {ast->pos.first - other.pos.first, ast->pos.second - other.pos.second};
		float distance = sqrt(distance_dir.x * distance_dir.x + distance_dir.y * distance_dir.y); //Pythagoras r = sqrt(x² + y²)
		
		// Bei Distanz gegen 0 geht Kraft gegen unendlich
		// -> Kraft begrenzen durch Softening Faktor EPS
		float soften_dist = sqrt(distance * distance + EPS * EPS);
		float acceleration = -1.0 * BIG_G * other.mass / (soften_dist * soften_dist);
	       	
		float2 distance_dir_unit = {distance_dir.x / soften_dist, distance_dir.y / soften_dist}; //Distance Vector normalisiert (durch Länge geteilt)

		acc.x += acceleration * distance_dir_unit.x;
		acc.y += acceleration * distance_dir_unit.y;	
	}

	__syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
                        //die späteren Blocke mit veränderten Daten arbeiten


	ast->velocity.first += acc.x * dt;
	ast->velocity.second += acc.y * dt;

	//printf("wert der draufkommt: %f, %f\n", ast->velocity.first * dt, ast->velocity.second * dt);
	ast->pos.first += ast->velocity.first * dt;
	ast->pos.second += ast->velocity.second * dt;

}	


void call_kernel(std::vector<Asteroid>& asteroids)
{
	if(asteroids.empty()) {
		return;
	}
    Asteroid* a = asteroids.data();
    Asteroid* d_asteroid;
     	
    int size = sizeof(Asteroid) * asteroids.size();

    hipMalloc(&d_asteroid, size);

    hipMemcpy(d_asteroid, a,size, hipMemcpyHostToDevice);

    calculate_forces<<<1, asteroids.size()>>>(d_asteroid, 1, asteroids.size());

    hipMemcpy(a, d_asteroid, size, hipMemcpyDeviceToHost);

    hipFree(d_asteroid);
}
