#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"
#include <thrust/device_vector.h>

void __global__ calculate_forces(Asteroid* d_a, float dt) 	//DelteTime ist die Zeit die zwischen zwei Berechnungen vergeht. Mussen irgendwie
{								//in der MainLoop dafür die Zeit stoppen und hier übergeben
	float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
	int astId = blockIdx.x * blockDim.x + threadIdx.x;
	Asteroid ast = d_a[astId];
	float2 acc = {0,0}; //Acceleration in x and y direction

	for(int i = 0; i < 4 /* asteroids.size() einfach übergeben?? */; i++){
		if(i == astId) continue; //Keine Gravity zu sich selber

		Asteroid other = d_a[i];
		float3 distance_dir = {ast.pos.first - other.pos.first, ast.pos.second - other.pos.second};
		float distance = sqrt(distance_dir.x * distance_dir.x + distance_dir.y * distance_dir.y); //Pythagoras r = sqrt(x² + y²)
		float acceleration = -1.0 * BIG_G * other.mass / (distance * distance);
	       	float2 distance_dir_unit = {distance_dir.x / distance, distance_dir.y / distance}; //Distance Vector normalisiert (durch Länge geteilt)

		acc.x += acceleration * distance_dir_unit.x;
		acc.y += acceleration * distance_dir_unit.y;	
	}

	__syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
                        //die späteren Blocke mit veränderten Daten arbeiten


	ast.velocity.first = acc.x * dt;
	ast.velocity.second = acc.y * dt;

	ast.pos.first = ast.velocity.first * dt;
	ast.pos.second = ast.velocity.second * dt;

}	


float call_kernel(std::vector<Asteroid>& asteroids)
{
    Asteroid* a = asteroids.data();
    Asteroid* d_asteroid;
     	
    int size = sizeof(Asteroid) * asteroids.size();

    hipMalloc(&d_asteroid, size);

    hipMemcpy(d_asteroid, a,size, hipMemcpyHostToDevice);

    calculate_forces<<<1, asteroids.size()>>>(d_asteroid, 0.1);  //Wollen wir hier den Pointer oder direkt einen Vector übergeben??

    hipMemcpy(a, d_asteroid, size, hipMemcpyDeviceToHost);
	    
    
}
