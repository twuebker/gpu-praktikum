#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"

__global__ void calculate_forces(Asteroid *asteroids) {
	
}

float call_kernel() {
	int n = 4;
	float* h_p = (float*)malloc(4 * sizeof(float) * n);
	float* h_a = (float*)malloc(3 * sizeof(float) * n);

	float* d_p;
	float* d_a;
	hipMalloc(&d_p, 4 * sizeof(float) * n);
	hipMalloc(&d_a, 4 * sizeof(float) * n);

	
}
