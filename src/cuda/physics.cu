#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"
#include <thrust/device_vector.h>

__global__ calculate_forces(Asteroid* d_a) {
	int astId = blockIdx.x * blockDim.x + threadIdx.x;
	
}	


float call_kernel(std::vector<Asteroid>& asteroids)
{
    Asteroid* a = asteroids.data();
    Asteroid* d_asteroid;
     	

    hipMalloc(&d_asteroid, sizeof(Asteroid) * asteroids.size());

    hipMemcpy(d_asteroid, a, sizeof(Asteroid) * asteroids.size(), hipMemcpyHostToDevice);

    calculate_forces<<<1, n>>>(d_asteroids);

    hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_a, d_a, sizeA, hipMemcpyDeviceToHost);

    std::cout << "Ergebnis: " << std::endl;
    for (int i = 0; i<4; i++)
    {
        std::cout << h_p[i] << " " << h_a[i] << std::endl;
    }
}
