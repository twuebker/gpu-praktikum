#include "hip/hip_runtime.h"
#define EPS (10)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <thrust/device_vector.h>
#include <stdio.h>

void __global__ calculate_forces(Asteroid* d_a, float dt, int size, ForceField* d_f, int sizeFF) 	//DelteTime ist die Zeit die zwischen zwei Berechnungen vergeht. Mussen irgendwie
{	
	//in der MainLoop dafür die Zeit stoppen und hier übergeben
	float BIG_G = 9.81; //Gravitationkonstante, aber am Ende voll abhängig wie groß unsere Zahlen so sind
	int astId = blockIdx.x * blockDim.x + threadIdx.x;
	Asteroid* ast = &d_a[astId];
	float2 acc = {0,0}; //Acceleration in x and y direction

	for(int i = 0; i < size; i++){
		if(i == astId) continue; //Keine Gravity zu sich selber

		Asteroid other = d_a[i];
		float2 distance_dir = {ast->pos.first - other.pos.first, ast->pos.second - other.pos.second};
		float distance = sqrt(distance_dir.x * distance_dir.x + distance_dir.y * distance_dir.y); //Pythagoras r = sqrt(x² + y²)
		
		// Bei Distanz gegen 0 geht Kraft gegen unendlich
		// -> Kraft begrenzen durch Softening Faktor EPS
		float soften_dist = sqrt(distance * distance + EPS * EPS);
		float acceleration = -1.0 * BIG_G * other.mass / (soften_dist * soften_dist);
	       	
		float2 distance_dir_unit = {distance_dir.x / soften_dist, distance_dir.y / soften_dist}; //Distance Vector normalisiert (durch Länge geteilt)

		acc.x += acceleration * distance_dir_unit.x;
		acc.y += acceleration * distance_dir_unit.y;	
	}
	
	for(int i = 0; i < sizeFF; i++) {
		ForceField force = d_f[i];
		if(!(ast->pos.first >= force.leftCorner.first && ast->pos.first <= force.rightCorner.first && ast->pos.second >= force.leftCorner.second && ast->pos.second <= force.rightCorner.second)) {
			continue;
		}
		switch(force.dir) {
				case Direction::UP:
						acc.y -= force.force /  ast->mass;
						break;
				case Direction::DOWN:
						acc.y += force.force / ast->mass;
						break;
				case Direction::LEFT:
						acc.x -= force.force / ast->mass;
						break;
				case Direction::RIGHT:
						acc.x += force.force / ast-> mass;		
						break;
				default:
					break;		
		}
	}
	__syncthreads(); //Synchronisiert alle Threads im Block. Können also so doch nur einen Block haben da sonst
                        //die späteren Blocke mit veränderten Daten arbeiten


	ast->velocity.first += acc.x * dt;
	ast->velocity.second += acc.y * dt;

	//printf("wert der draufkommt: %f, %f\n", ast->velocity.first * dt, ast->velocity.second * dt);
	ast->pos.first += ast->velocity.first * dt;
	ast->pos.second += ast->velocity.second * dt;

}	

void call_kernel(std::vector<Asteroid>& asteroids, std::vector<ForceField>& forceFields)
{
	std::cout << "Begin" << std::endl;
	if(asteroids.empty()) {
		return;
	}
    Asteroid* a = asteroids.data();
    Asteroid* d_asteroid;

    ForceField* f = forceFields.data();
    ForceField* d_forceField;
    std::cout << "MOIN" << std::endl;
    int size = sizeof(Asteroid) * asteroids.size();
	int sizeFF = sizeof(ForceField) * forceFields.size();
    hipMalloc(&d_asteroid, size);
	hipMalloc(&d_forceField, sizeFF);
	
    hipMemcpy(d_asteroid, a,size, hipMemcpyHostToDevice);
	hipMemcpy(d_forceField, f, sizeFF, hipMemcpyHostToDevice);
	std::cout << "HALLO" << std::endl;
    calculate_forces<<<1, asteroids.size()>>>(d_asteroid, 0.1, asteroids.size(), d_forceField, forceFields.size());
	std::cout << "ALLES GECALCT" << std::endl;
    hipMemcpy(a, d_asteroid, size, hipMemcpyDeviceToHost);
	std::cout << "HI" << std::endl;
    hipFree(d_asteroid);
	hipFree(d_forceField);
	std::cout << "FERTIG" << std::endl;
}
