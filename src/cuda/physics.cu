#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"

__device__ float3 tile_calculation(float4 myPosition, float3 accel)
{
    int i;
    extern __shared__ float4[] shPosition;
    for (i = 0; i < blockDim.x; i++) 
    {
        accel = bodyBodyInteraction(myPosition, shPosition[i], accel);
    }
    return accel; 
}

__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai) 
{
    float3 r;
       // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
       r.y = bj.y - bi.y;
       r.z = bj.z - bi.z;
       // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
        float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;
       // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
        float distSixth = distSqr * distSqr * distSqr;
       float invDistCube = 1.0f/sqrtf(distSixth);
       // s = m_j * invDistCube [1 FLOP]
        float s = bj.w * invDistCube;
       // a_i =  a_i + s * r_ij [6 FLOPS]
       ai.x += r.x * s;
       ai.y += r.y * s;
    ai.z += r.z * s;
    return ai;
}

__global__ void calculate_forces(void *devX, void *devA)
{
	extern __shared__ float4[] shPosition;
	float4 *globalX = (float4 *)devX;
	float4 *globalA = (float4 *)devA;
	float4 myPosition;
	int i, tile;
	int p = 1;
	int N = 4;
	float3 acc = {0.0f, 0.0f, 0.0f};
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	myPosition = globalX[gtid];
	for(i=0, tile = 0; i < N; i += p, tile++)
	{
		int idx = tile* blockDim.x + threadIdx.x;
		shPosition[threadIdx.x] = globalX[idx];
		__syncthreads();
		acc = tile_calculation(myPosition, acc);
		__syncthreads();
	}
	float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
	globalA[gtid] = acc4;
}



float call_kernel() {
	int n = 4;
	int size = 4 * sizeof(float) * n;
	int sizeA = 3 * sizeof(float) * n;
	float* h_p = (float*)malloc(size);
	float* h_a = (float*)malloc(sizeA);


	for(int i = 0; i < 4; i++) {
		h_p[i] = i;
		h_a[i] = i;
	}
	float* d_p;
	float* d_a;
	hipMalloc(&d_p, size);
	hipMalloc(&d_a, sizeA);

	hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
	hipMemcpy(d_a, h_a, sizeA, hipMemcpyHostToDevice);
	
	calculate_forces<<<1,n,n>>>(d_p, d_a);

	hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_a, d_a, sizeA, hipMemcpyDeviceToHost);

	std::cout << "Ergebnis: " << std::endl;
	for(int i = 0; i < 4; i++) {
		std::cout << h_p[i] << " " << h_a[i] << std::endl;
	}
}
