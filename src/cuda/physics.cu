#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../widgets/AsteroidsScene.h"

__global__ void calculate_forces(void *devX, void *devA)
{
	extern __shared__ float4[] shPosition;
	float4 *globalX = (float4 *)devX;
	float4 *globalA = (float4 *)devA;
	float4 myPosition;
	int i, tile;
	float3 acc = {0.0f, 0.0f, 0.0f};
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	myPosition = globalX[gtid];
	for(i=0, tile = 0; i < N; i += p, tile++)
	{
		int idx = tile* blockDim.x + threadIdx.x;
		shPosition[threadIdx.x] = globalX[idx];
		__syncthreads();
		acc = tile_calculation(myPosition, acc);
		__syncthreads();
	}
	float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
	globalA[gtid] = acc4;
}



float call_kernel() {
	int n = 4;
	int size = 4 * sizeof(float) * n;
	float* h_p = (float*)malloc(size);
	float* h_a = (float*)malloc(size);

	float* d_p;
	float* d_a;
	hipMalloc(&d_p, size);
	hipMalloc(&d_a, size);

	hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

}
